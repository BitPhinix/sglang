#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>
#include <c10/util/irange.h>

#include <cstdint>

#include "pytorch_extension_utils.h"

__device__ __forceinline__ void
transfer_item_warp(int32_t lane_id, const void* src_addr, void* dst_addr, int64_t item_size_bytes) {
  // todo, different chunk size
  int total_chunks = item_size_bytes / 8;
  const int64_t* src_8 = reinterpret_cast<const int64_t*>(src_addr);
  int64_t* dst_8 = reinterpret_cast<int64_t*>(dst_addr);
#pragma unroll
  for (int j = lane_id; j < total_chunks; j += 32) {
    const int64_t* src_addr_lane = &src_8[j];
    int64_t* dst_addr_lane = &dst_8[j];
    int64_t temp_val;
    asm volatile("ld.global.nc.b64 %0, [%1];" : "=l"(temp_val) : "l"(src_addr_lane) : "memory");
    asm volatile("st.global.cg.b64 [%0], %1;" ::"l"(dst_addr_lane), "l"(temp_val) : "memory");
  }
}

// todo, structs for different memory layout
__device__ __forceinline__ int64_t
get_global_offset_lf(int64_t layer_id, int64_t layer_dim, int64_t page_id, int64_t item_size_bytes) {
  // layer first
  return layer_id * layer_dim + page_id * item_size_bytes;
}

__device__ __forceinline__ int64_t
get_global_offset_pf(int64_t layer_id, int64_t page_dim, int64_t page_id, int64_t item_size_bytes) {
  // page first
  return page_id * page_dim + layer_id * item_size_bytes;
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
__global__ void transfer_kernel_impl(
    const void* __restrict__ src_k,
    void* __restrict__ dst_k,
    const void* __restrict__ src_v,
    void* __restrict__ dst_v,
    const int64_t* __restrict__ src_indices,
    const int64_t* __restrict__ dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t num_items,
    int64_t items_per_warp,
    int64_t item_size_bytes,
    int64_t src_layout_dim,
    int64_t dst_layout_dim) {
  int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  int32_t lane_id = tid % 32;
  int32_t warp_id = tid / 32;

  for (int i = 0; i < items_per_warp; ++i) {
    int32_t item_id = warp_id * items_per_warp + i;
    if (item_id >= num_items) {
      return;
    }
    const int64_t src_page_id = src_indices[item_id];
    const int64_t dst_page_id = dst_indices[item_id];

    // Loop over layers if necessary
    for (int64_t layer_id = start_layer_id; layer_id < start_layer_id + num_layers_to_process; ++layer_id) {
      // Calculate offsets using the provided function pointers
      const int64_t src_offset = SrcOffsetFn(layer_id, src_layout_dim, src_page_id, item_size_bytes);
      const int64_t dst_offset = DstOffsetFn(layer_id, dst_layout_dim, dst_page_id, item_size_bytes);

      if constexpr (IsMLA) {
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_k) + src_offset,
            static_cast<char*>(dst_k) + dst_offset,
            item_size_bytes);
      } else {
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_k) + src_offset,
            static_cast<char*>(dst_k) + dst_offset,
            item_size_bytes);
        transfer_item_warp(
            lane_id,
            static_cast<const char*>(src_v) + src_offset,
            static_cast<char*>(dst_v) + dst_offset,
            item_size_bytes);
      }
    }
  }
}

template <auto SrcOffsetFn, auto DstOffsetFn, bool IsMLA>
void transfer_kv_launcher(
    const at::Tensor& src_k,
    at::Tensor& dst_k,
    const at::Tensor& src_v,
    at::Tensor& dst_v,
    const at::Tensor& src_indices,
    const at::Tensor& dst_indices,
    int64_t start_layer_id,
    int64_t num_layers_to_process,
    int64_t item_size,
    int64_t src_layout_dim,
    int64_t dst_layout_dim,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  TORCH_CHECK(src_k.scalar_type() == dst_k.scalar_type(), "Source and destination keys must have the same type");
  TORCH_CHECK(src_indices.is_cuda(), "Source indices must be a CUDA tensor");
  TORCH_CHECK(dst_indices.is_cuda(), "Destination indices must be a CUDA tensor");
  TORCH_CHECK(src_indices.scalar_type() == at::kLong, "Source indices must be of type long");
  TORCH_CHECK(dst_indices.scalar_type() == at::kLong, "Destination indices must be of type long");
  TORCH_CHECK(src_indices.numel() == dst_indices.numel(), "Source and destination indices must have the same length");

  if (!IsMLA) {
    TORCH_CHECK(src_v.scalar_type() == dst_v.scalar_type(), "Source and destination values must have the same type");
  }

  int dtype_size = src_k.element_size();
  TORCH_CHECK((item_size * dtype_size) % 8 == 0, "Item byte size must be divisible by 8");

  auto div_up = [](int32_t x, int32_t y) { return (x + y - 1) / y; };
  const int64_t num_items = src_indices.numel();
  const int64_t items_per_warp = div_up(num_items, block_quota * num_warps_per_block);
  const int32_t num_blocks = div_up(num_items, items_per_warp * num_warps_per_block);
  dim3 grid_dim(num_blocks, 1, 1);
  const int32_t threads_per_block = num_warps_per_block * 32;

  hipStream_t torch_current_stream = at::cuda::getCurrentCUDAStream();
  transfer_kernel_impl<SrcOffsetFn, DstOffsetFn, IsMLA><<<grid_dim, threads_per_block, 0, torch_current_stream>>>(
      src_k.data_ptr(),
      dst_k.data_ptr(),
      (IsMLA ? nullptr : src_v.data_ptr()),
      (IsMLA ? nullptr : dst_v.data_ptr()),
      src_indices.data_ptr<int64_t>(),
      dst_indices.data_ptr<int64_t>(),
      start_layer_id,
      num_layers_to_process,
      num_items,
      items_per_warp,
      item_size * dtype_size,
      src_layout_dim * dtype_size,
      dst_layout_dim * dtype_size);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void transfer_kv_per_layer(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, false>(
      src_k, dst_k, src_v, dst_v, src_indices, dst_indices, 0, 1, item_size, 0, 0, block_quota, num_warps_per_block);
}

void transfer_kv_all_layer(
    const at::Tensor src_k,
    at::Tensor dst_k,
    const at::Tensor src_v,
    at::Tensor dst_v,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t src_layer_offset,
    int64_t dst_layer_offset,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, false>(
      src_k,
      dst_k,
      src_v,
      dst_v,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      src_layer_offset,
      dst_layer_offset,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_per_layer_mla(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty_tensor = at::Tensor();
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, true>(
      src,
      dst,
      empty_tensor,
      empty_tensor,
      src_indices,
      dst_indices,
      0,
      1,
      item_size,
      0,
      0,
      block_quota,
      num_warps_per_block);
}

void transfer_kv_all_layer_mla(
    const at::Tensor src,
    at::Tensor dst,
    const at::Tensor src_indices,
    const at::Tensor dst_indices,
    int64_t item_size,
    int64_t num_layers,
    int64_t src_layer_offset,
    int64_t dst_layer_offset,
    int64_t block_quota,
    int64_t num_warps_per_block) {
  at::Tensor empty_tensor = at::Tensor();
  transfer_kv_launcher<get_global_offset_lf, get_global_offset_lf, true>(
      src,
      dst,
      empty_tensor,
      empty_tensor,
      src_indices,
      dst_indices,
      0,
      num_layers,
      item_size,
      src_layer_offset,
      dst_layer_offset,
      block_quota,
      num_warps_per_block);
}

inline void transfer_page_direct(
    at::Tensor src_buffer,
    at::Tensor dst_buffer,
    int64_t src_page_start_index,
    int64_t dst_page_start_index,
    int64_t page_size) {
  dst_buffer.slice(0, dst_page_start_index, dst_page_start_index + page_size)
      .copy_(
          src_buffer.slice(0, src_page_start_index, src_page_start_index + page_size),
          /* non_blocking= */ true);
}

void transfer_kv_to_cpu_all_layer_direct(
    at::Tensor host_indices,
    at::Tensor host_k_buffer,
    at::Tensor host_v_buffer,
    at::Tensor device_indices,
    at::Tensor device_k_buffer,
    at::Tensor device_v_buffer,
    int64_t page_size,
    int64_t layer_num) {
  TORCH_CHECK(
      host_indices.numel() == device_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(host_indices.numel() % page_size == 0, "Source indice size must be divisible by page size");

  auto device_indices_host = device_indices.cpu();
  for (const auto i : c10::irange(device_indices_host.size(0) / page_size)) {
    auto h_index = host_indices[i * page_size].item<int64_t>();
    auto d_index = device_indices_host[i * page_size].item<int64_t>();
    for (const auto j : c10::irange(layer_num)) {
      transfer_page_direct(device_k_buffer[j], host_k_buffer[j], d_index, h_index, page_size);
      transfer_page_direct(device_v_buffer[j], host_v_buffer[j], d_index, h_index, page_size);
    }
  }
}

void transfer_kv_to_gpu_per_layer_direct(
    at::Tensor host_indices,
    at::Tensor host_k_buffer,
    at::Tensor host_v_buffer,
    at::Tensor device_indices,
    at::Tensor device_k_buffer,
    at::Tensor device_v_buffer,
    int64_t page_size,
    int64_t layer_id) {
  TORCH_CHECK(
      host_indices.numel() == device_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(host_indices.numel() % page_size == 0, "Source indice size must be divisible by page size");

  auto device_indices_host = device_indices.cpu();
  for (const auto i : c10::irange(device_indices_host.size(0) / page_size)) {
    auto h_index = host_indices[i * page_size].item<int64_t>();
    auto d_index = device_indices_host[i * page_size].item<int64_t>();
    transfer_page_direct(host_k_buffer[layer_id], device_k_buffer[layer_id], h_index, d_index, page_size);
    transfer_page_direct(host_v_buffer[layer_id], device_v_buffer[layer_id], h_index, d_index, page_size);
  }
}

void transfer_kv_to_cpu_all_layer_direct_mla(
    at::Tensor host_indices,
    at::Tensor host_buffer,
    at::Tensor device_indices,
    at::Tensor device_buffer,
    int64_t page_size,
    int64_t layer_num) {
  TORCH_CHECK(
      host_indices.numel() == device_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(host_indices.numel() % page_size == 0, "Source indice size must be divisible by page size");

  auto device_indices_host = device_indices.cpu();
  for (const auto i : c10::irange(device_indices_host.size(0) / page_size)) {
    auto h_index = host_indices[i * page_size].item<int64_t>();
    auto d_index = device_indices_host[i * page_size].item<int64_t>();
    for (const auto j : c10::irange(layer_num)) {
      transfer_page_direct(device_buffer[j], host_buffer[j], d_index, h_index, page_size);
    }
  }
}

void transfer_kv_to_gpu_per_layer_direct_mla(
    at::Tensor host_indices,
    at::Tensor host_buffer,
    at::Tensor device_indices,
    at::Tensor device_buffer,
    int64_t page_size) {
  TORCH_CHECK(
      host_indices.numel() == device_indices.numel(), "Source and destination indices must have the same length");
  TORCH_CHECK(host_indices.numel() % page_size == 0, "Source indice size must be divisible by page size");

  auto device_indices_host = device_indices.cpu();
  for (const auto i : c10::irange(device_indices_host.size(0) / page_size)) {
    auto h_index = host_indices[i * page_size].item<int64_t>();
    auto d_index = device_indices_host[i * page_size].item<int64_t>();
    transfer_page_direct(host_buffer, device_buffer, h_index, d_index, page_size);
  }
}
